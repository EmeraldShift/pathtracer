#include "hip/hip_runtime.h"
#include "tracer.h"

#include "scene/scene.h"

#include <atomic>
#include <mutex>
#include <random>
#include <iostream>
#include <glm/gtx/norm.hpp>
#include <glm/gtx/io.hpp>

// For use in threading
struct WorkUnit {
    WorkUnit *next;
    int minX, maxX;
    int minY, maxY;
};
std::mutex l;
int totalWork;
WorkUnit *head = nullptr;
std::atomic<int> workDone;
std::atomic<int> threadsDone;

template<typename Numeric, typename Generator = std::mt19937>
static Numeric random(Numeric from, Numeric to) {
    thread_local static Generator gen(std::random_device{}());

    using dist_type = typename std::conditional<std::is_integral<Numeric>::value,
            std::uniform_int_distribution<Numeric>,
            std::uniform_real_distribution<Numeric>>::type;

    thread_local static dist_type dist;

    return dist(gen, typename dist_type::param_type{from, to});
}

static std::pair<f4, f4> getBasis(f4 normal) {
    auto a = std::abs(normal[0]) > RAY_EPSILON
             ? f4m::normalize(f4m::cross({0, 1, 0}, normal))
             : f4m::normalize(f4m::cross({1, 0, 0}, normal));
    auto b = f4m::cross(normal, a);
    return std::make_pair(a, b);
}

static f4 randomVecFromHemisphere(f4 normal) {
    auto basis = getBasis(normal);
    float p = 2 * 3.141592f * random<float>(0, 1);
    float cos_p = std::cos(p);
    float sin_p = std::sin(p);
    float cos_t = std::pow(random<float>(0, 1), 2);
    float sin_t = std::sqrt(1.0f - cos_t * cos_t);
    return sin_t * cos_p * basis.first + sin_t * sin_p * basis.second + cos_t * normal;
}

constexpr int granularity = 32;

void CpuTracer::traceImage(int w, int h) {
    for (int y = 0; y < h; y += granularity) {
        for (int x = 0; x < w; x += granularity) {
            auto *work = new WorkUnit;
            work->minX = x;
            work->minY = y;
            work->maxX = std::min(x + granularity, w);
            work->maxY = std::min(y + granularity, h);
            work->next = head;
            head = work;
            totalWork++;
        }
    }

    for (unsigned i = 0; i < threads; i++) {
        workers[i] = new std::thread([&, i] {
            while (true) {
                WorkUnit *work;
                l.lock();
                if (head) {
                    work = head;
                    head = head->next;
                    l.unlock();
                } else {
                    l.unlock();
                    break;
                }
                for (int y = work->minY; y < work->maxY; y++)
                    for (int x = work->minX; x < work->maxX; x++)
                        setPixel(x, y, tracePixel(x, y));
                delete work;

                // Announce progress
                int done;
                if ((done = ++workDone) % (std::max(1, totalWork / 20)) == 0)
                    std::cout << done * 100 / totalWork << "%" << std::endl;
            }
            threadsDone++;
        });
    }
}

f4 CpuTracer::tracePixel(int i, int j) {
    f4 sum;
    for (auto x = 0; x < samples; x++) {
        for (auto y = 0; y < samples; y++) {
            auto xx = float(i) - 0.5f + (1 + 2 * x) / (2.0f * samples);
            auto yy = float(j) - 0.5f + (1 + 2 * y) / (2.0f * samples);
            sum += trace(xx / float(buffer_width), yy / float(buffer_height));
        }
    }
    return sum / ((float) samples * samples);
}

f4 CpuTracer::trace(float x, float y) {
    ray r;
    scene->getCamera().rayThrough(x, y, r);
    f4 ret = traceRay(r, 1, depth);
    ret = f4m::clamp(ret, 0.0f, 1.0f);
    return ret;
}

f4 CpuTracer::traceRay(ray &r, const f4 &thresh, int depth) {
    isect i;
    if (f4m::length2(thresh) < 3.0 * 12.0 / 255.0 / 255.0)
        return 0;

    if (!scene->intersect(r, i))
        return cubeMap ? cubeMap->getColor(r) : 1;

    auto hitInner = r.getPosition() + (i.getT() + RAY_EPSILON) * r.getDirection();
    auto hitOuter = r.getPosition() + (i.getT() - RAY_EPSILON) * r.getDirection();

    if (depth < 0)
        return i.getMaterial().ke(i) * 32.0f;

    f4 rad;

    auto diffuse = f4m::length(i.getMaterial().kd(i));
    auto reflect = i.getMaterial().kr(i)[0];
    auto refract = i.getMaterial().kt(i)[0];
    auto rand = random<float>(0, 1);

    auto refl = r.getDirection() - 2.0f * i.getN() * f4m::dot(i.getN(), r.getDirection());
    if (rand < refract) {
        bool into = f4m::dot(r.getDirection(), i.getN()) < 0;
        auto n = into ? i.getN() : -i.getN();
        auto n1 = 1.0f;
        auto n2 = i.getMaterial().index(i);
        auto ratio = into ? n1 / n2 : n2 / n1;
        auto dot = f4m::dot(r.getDirection(), n);
        auto cos2t = 1 - ratio * ratio * (1 - dot * dot);
        if (cos2t < 0) {
            ray rr(hitOuter, refl);
            auto w = i.getMaterial().kr(i) / 1.0f;
            rad += w * traceRay(rr, w * thresh, depth - 1);
        } else {
            auto dir = f4m::normalize(r.getDirection() * ratio - n * (dot * ratio + std::sqrt(cos2t)));
            ray rr(hitInner, dir);
            auto a = n2 - n1;
            auto b = n2 + n1;
            auto R0 = (a * a) / (b * b);
            auto c = 1.0f - (into ? -dot : f4m::dot(rr.getDirection(), -n));
            auto Re = R0 + (1.0f - R0) * c * c * c * c * c;
            auto ratio2 = ratio * ratio;
            auto Tr = (1.0f - Re) * ratio2;

            auto prob = 0.25f + 0.5f * Re;
            // XXX depth test
            rad = Tr * traceRay(rr, Tr * thresh, depth - 1);
        }
    } else {
        auto diff = randomVecFromHemisphere(i.getN());
        auto dir = f4m::normalize(reflect * refl + (1 - reflect) * diff);
        ray rr(hitOuter, dir);
//        auto color = i.getMaterial().kd(i) + glm::vec3(0.15, 0.15, 0.15); // Makes even black a little reflective
        auto w = i.getMaterial().kd(i) / 1.0f;
        rad = w * (traceRay(rr, w * thresh, depth - 1));
//        auto w = glm::sqrt(color) / 1.0;
//        rad = w * traceRay(rr, w * thresh, depth - 1);
    }
//    else {
//        auto basis = getBasis(i.getN());
//        auto dir = randomVecFromHemisphere(i.getN());
//        ray rr(hitOuter, dir);
//        auto w = glm::vec3(1) / 1.0;
//        rad = w * (i.getMaterial().kd(i) + traceRay(rr, w * thresh, depth - 1)) / 2.0;
//    }
    return i.getMaterial().ke(i) * 32.0f + rad;
}

void CpuTracer::waitRender() {
    for (unsigned i = 0; i < threads; i++)
        workers[i]->join();
}