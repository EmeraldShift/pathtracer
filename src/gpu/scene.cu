#include "../scene/scene.h"

Scene *Scene::clone() const {
    Scene *d_scene;
    hipMalloc(&d_scene, sizeof(Scene));
    hipMemcpy(d_scene, this, sizeof(Scene), hipMemcpyHostToDevice);

    // Clone BVH
    BoundedVolumeHierarchy d_bvh = bvh.clone();
    hipMemcpy(&d_scene->bvh, &d_bvh, sizeof(BoundedVolumeHierarchy), hipMemcpyHostToDevice);
    return d_scene;
}