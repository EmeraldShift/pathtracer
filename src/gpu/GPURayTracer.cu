#include "../RayTracer.h"
#include "cuda.cuh"

void RayTracer::traceImageGPU(int w, int h) {
    //sets up buffers, updates ui parameters
    traceSetup(w, h);

    //copy pixel buffer onto device
    //hipMalloc
    //hipMemcpy

    //determine kenel paramaters
    int num_blocks = 1;
    int block_size = 1;

    //launch tracer kernels
    cuda_hello<<<num_blocks, block_size>>>();

    //sync
    hipDeviceSynchronize();
}

