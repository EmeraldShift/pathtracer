#include "../RayTracer.h"
#include "cuda.cuh"

void RayTracer::traceImageGPU(int w, int h) {
    //sets up buffers, updates ui parameters
    traceSetup(w, h);

    //copy pixel buffer onto device
    //hipMalloc
    //hipMemcpy

    //determine kenel paramaters
    int num_blocks = 1;
    int block_size = 1;

    //launch tracer kernels
    cuda_hello<<<num_blocks, block_size>>>();

    //render
}


void RayTracer::waitRender() {
    if (m_gpu)
        hipDeviceSynchronize();
    else
        for (int i = 0; i < threads; i++)
            workers[i]->join();

}