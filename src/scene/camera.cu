#include "hip/hip_runtime.h"
#include <iostream>
#include "camera.h"
#include "../ui/TraceUI.h"

#define PI 3.14159265359
#define SHOW(x) (cerr << #x << " = " << (x) << "\n")

void Camera::rayThrough(double x, double y, ray &r) const
// Ray through normalized window point x,y.  In normalized coordinates
// the camera's x and y vary both vary from 0 to 1.
{
    x -= 0.5;
    y -= 0.5;
    glm::dvec3 dir = glm::normalize(look + x * u + y * v);
    r.setPosition(eye);
    r.setDirection(dir);
}

void Camera::setEye(const glm::dvec3 &eye) {
    this->eye = eye;
}

void Camera::setLook(double r, double i, double j, double k)
{
    m[0][0] = 1.0 - 2.0 * (i * i + j * j);
    m[0][1] = 2.0 * (r * i - j * k);
    m[0][2] = 2.0 * (j * r + i * k);

    m[1][0] = 2.0 * (r * i + j * k);
    m[1][1] = 1.0 - 2.0 * (j * j + r * r);
    m[1][2] = 2.0 * (i * j - r * k);

    m[2][0] = 2.0 * (j * r - i * k);
    m[2][1] = 2.0 * (i * j + r * k);
    m[2][2] = 1.0 - 2.0 * (i * i + r * r);

    m = glm::transpose(m);
    update();
}

void
Camera::setLook(const glm::dvec3 &viewDir, const glm::dvec3 &upDir) {
    glm::dvec3 z = -viewDir;
    const glm::dvec3 &y = upDir;
    glm::dvec3 x = glm::cross(y, z);
    m = glm::dmat3x3(x, y, z); // Do we need to transpose?
    update();
}

void
Camera::setFOV(double fov)
{
    fov /= (180.0 / PI);
    normalizedHeight = 2 * tan(fov / 2);
    update();
}

void
Camera::setAspectRatio(double ar)
{
    aspectRatio = ar;
    update();
}

void
Camera::update() {
    u = m * glm::dvec3(1, 0, 0) * normalizedHeight * aspectRatio;
    v = m * glm::dvec3(0, 1, 0) * normalizedHeight;
    look = m * glm::dvec3(0, 0, -1);
}
