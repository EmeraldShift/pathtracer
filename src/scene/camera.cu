#include "hip/hip_runtime.h"
#include <iostream>
#include "camera.h"
#include "../ui/TraceUI.h"

#define PI 3.14159265359
#define SHOW(x) (cerr << #x << " = " << (x) << "\n")

__host__ __device__
void Camera::rayThrough(float x, float y, ray &r) const
// Ray through normalized window point x,y.  In normalized coordinates
// the camera's x and y vary both vary from 0 to 1.
{
    x -= 0.5;
    y -= 0.5;
    glm::vec3 dir = glm::normalize(look + x * u + y * v);
    r.setPosition(eye);
    r.setDirection(dir);
}

void Camera::setEye(const glm::vec3 &eye) {
    this->eye = eye;
}

void Camera::setLook(float r, float i, float j, float k)
{
    m[0][0] = 1.0 - 2.0 * (i * i + j * j);
    m[0][1] = 2.0 * (r * i - j * k);
    m[0][2] = 2.0 * (j * r + i * k);

    m[1][0] = 2.0 * (r * i + j * k);
    m[1][1] = 1.0 - 2.0 * (j * j + r * r);
    m[1][2] = 2.0 * (i * j - r * k);

    m[2][0] = 2.0 * (j * r - i * k);
    m[2][1] = 2.0 * (i * j + r * k);
    m[2][2] = 1.0 - 2.0 * (i * i + r * r);

    m = glm::transpose(m);
    update();
}

void
Camera::setLook(const glm::vec3 &viewDir, const glm::vec3 &upDir) {
    glm::vec3 z = -viewDir;
    const glm::vec3 &y = upDir;
    glm::vec3 x = glm::cross(y, z);
    m = glm::dmat3x3(x, y, z); // Do we need to transpose?
    update();
}

void
Camera::setFOV(float fov)
{
    fov /= (180.0 / PI);
    normalizedHeight = 2 * tan(fov / 2);
    update();
}

void
Camera::setAspectRatio(float ar)
{
    aspectRatio = ar;
    update();
}

void
Camera::update() {
    u = m * glm::vec3(1, 0, 0) * normalizedHeight * aspectRatio;
    v = m * glm::vec3(0, 1, 0) * normalizedHeight;
    look = m * glm::vec3(0, 0, -1);
}
