#include "geometry.h"

Geometry *Geometry::clone() const {
    Geometry *d_geom;
    hipMalloc(&d_geom, sizeof(Geometry));
    hipMemcpy(d_geom, this, sizeof(Geometry), hipMemcpyHostToDevice);
    return d_geom;
}