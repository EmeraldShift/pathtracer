#include "hip/hip_runtime.h"
#include "material.h"
#include "../ui/TraceUI.h"
#include "ray.h"

#include <algorithm>
#include "../fileio/images.h"

TextureMap::TextureMap(const string& filename) {
    data = readImage(filename.c_str(), width, height);
    if (data.empty()) {
        width = 0;
        height = 0;
        string error("Unable to load texture map '");
        error.append(filename);
        error.append("'.");
        throw TextureMapException(error);
    }
}

f4 TextureMap::getMappedValue(const float2 &coord) const {
    auto x = coord.x * width;
    auto y = coord.y * height;

    auto lerp_x = x - (int) x;
    auto lerp_y = y - (int) y;

    auto vlerp_xl = lerp_x * getPixelAt((int) x + 1, (int) y) + (1 - lerp_x) * getPixelAt((int) x, (int) y);
    auto vlerp_xr = lerp_x * getPixelAt((int) x + 1, (int) y + 1) + (1 - lerp_x) * getPixelAt((int) x, (int) y + 1);
    return lerp_y * vlerp_xr + (1 - lerp_y) * vlerp_xl;
}

f4 TextureMap::getPixelAt(int x, int y) const {
    x = std::max(0, std::min(width - 1, x));
    y = std::max(0, std::min(height - 1, y));
    auto idx = (x + y * width) * 3;
    return {data[idx] / 256.0f, data[idx + 1] / 256.0f, data[idx + 2] / 256.0f};
}
